#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"
 //static __device__ __inline__ float fresnel(float cos_theta_i, float cos_theta_t, float eta);


//rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(PerRayData_beam, prd_beam, rtPayload, );

rtDeclareVariable(optix::Ray,  ray,          rtCurrentRay, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float, refraction_index, , );
//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        transducer_eye, , );
rtDeclareVariable(float3,		   transducer_U, , );
rtDeclareVariable(float3,		   transducer_V, , );
rtDeclareVariable(float3,		   transducer_W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;


using namespace optix;

RT_PROGRAM void ray_generation()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;

	float3 ray_origin = transducer_eye;
	//float3 ray_origin2 = make_float3(-5.0f, 60.0f, -16.0f);
	//float3 ray_direction = normalize(d.x*transducer_U + d.y*transducer_V + transducer_W);
	float3 ray_direction = normalize(make_float3(0.0f, 4.0f, 0.0f) - ray_origin);
	Ray ray(ray_origin, ray_direction, BEAM_RAY_TYPE, scene_epsilon);


	PerRayData_beam first_prd;
	first_prd.depth = 0;
	rtTrace(top_object, ray, first_prd);
	rtPrintf("1: [%f, %f, %f]\n", first_prd.hit_pos.x, first_prd.hit_pos.y, first_prd.hit_pos.z);

	PerRayData_beam second_prd;
	float3 R1;
	rtPrintf("/////////////// [%f, %f, %f]\n", first_prd.ffnormal.x, first_prd.ffnormal.y, first_prd.ffnormal.z);
	if (refract(R1, normalize(first_prd.ray_direction), first_prd.ffnormal, refraction_index))
	{ //1.4f
		Ray refraction1(first_prd.hit_pos, R1, BEAM_RAY_TYPE, scene_epsilon);
		rtTrace(top_object, refraction1, second_prd);
	}
	//rtprintf("2:  [%f, %f, %f]\n", r1.x, r1.y, r1.z);
	rtPrintf("\n\n 2:  [%f, %f, %f]\n", second_prd.hit_pos.x, second_prd.hit_pos.y, second_prd.hit_pos.z);


	PerRayData_beam third_prd;
	float3 R2;
	if (refract(R2, normalize(second_prd.ray_direction), second_prd.ffnormal, refraction_index)) {
		Ray refraction2(second_prd.hit_pos, R2, BEAM_RAY_TYPE, scene_epsilon);
		rtTrace(top_object, refraction2, third_prd);
	}
	//rtPrintf("2:  [%f, %f, %f]\n", R2.x, R2.y, R2.z);
	rtPrintf("\n\n 3: [%f, %f, %f]\n", third_prd.hit_pos.x, third_prd.hit_pos.y, third_prd.hit_pos.z);
}


RT_PROGRAM void closest_hit_beam()
{
//  prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;


	prd_beam.ray_direction = ray.direction;
	prd_beam.ffnormal = ffnormal;
	prd_beam.hit_pos = hit_point;
	//prd_beam.depth = prd_beam.depth + 1;
}

RT_PROGRAM void any_hit()
{
	rtTerminateRay();
}
 
//RT_PROGRAM void miss()
//{
//	rtPrintf("%d: miss\n", prd_beam.depth);
//}



//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{

	rtPrintExceptionDetails();
 //output_buffer[launch_index] = make_color( bad_color );
}
