#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "raytracing.h"
#include <hip/hip_complex.h>
#define HIP_PI_F 3.141592654f


//ray
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(PerRayData_beam, prd_beam, rtPayload, );
rtDeclareVariable(optix::Ray,  ray,          rtCurrentRay, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float3, target, , );


//properties
rtDeclareVariable(float, water_density, , );
rtDeclareVariable(float, water_speed, , );
rtDeclareVariable(float, skull_density, , );
rtDeclareVariable(float, skull_speed, , );
rtDeclareVariable(float, random_density, , );
rtDeclareVariable(float, random_speed, , );

//output buffer
rtBuffer<double, 2> out_rc_buffer;
rtBuffer<double, 2> test_rc_buffer;
rtBuffer<float3, 2> out_intersection_point;
rtBuffer<float3, 2> in_intersection_point;
rtBuffer<float3, 2> reflec_in_skull2out;

//input buffer
rtBuffer<float3>        origins;

using namespace optix;

__device__ __forceinline__ hipDoubleComplex _exp(hipDoubleComplex z)
{
	hipDoubleComplex res;
	float t = expf(z.x);
	sincos(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;
	return res;
}

RT_PROGRAM void ray_generation()
{
	float lambda = 1000 * skull_speed / 200000;
	float k = 2 * HIP_PI_F / lambda;

	double angle_outskull = 0.0;
	double angle_inskull = 0.0;
	double angle_reflection = 0.0;
	double out_rc;
	double in_rc;
	double reflect_rc;
	double test_rc;
	double RC_water_skull;
	double TC_water_skull;
	double RC_skull_brain;
	double TC_skull_brain;
	double RC_skull_water;
	double TC_skull_water;
	float3 in_out_dist;
	float3 in_reflection_dist;
	double travel_length;
	double travel_length_reflection;
	hipDoubleComplex phase_refraction;
	hipDoubleComplex phase_reflection;
	double out_rc_sum = 0;
	double test_rc_sum = 0;
	double ARC;
	double Out_ARC;



	int idx = launch_index.x + launch_index.y;

		float3 ray_origin = origins[idx];
		float3 ray_direction = normalize(target - ray_origin);

		////////////////////////////////////////////
		//first ray:: transducer -> skull layer1
		///////////////////////////////////////////
		Ray ray(ray_origin, ray_direction, BEAM_RAY_TYPE, scene_epsilon);
		//rtPrintf("%d ray_origin == : [%f %f %f]\n", idx, ray_origin.x, ray_origin.y, ray_origin.x);
		PerRayData_beam first_prd;
		first_prd.isHit = false;
		first_prd.isDone = false;
		rtTrace(top_object, ray, first_prd);

		float3 out_skull_entervector;
		////////////////////////////////////////////
		//out_rc:: transducer -> skull layer1	
		///////////////////////////////////////////
		if (first_prd.isHit) {

			out_skull_entervector = target - ray_origin;
			angle_outskull = getIncidenceAngle(out_skull_entervector, first_prd.ffnormal);
			out_rc = reflection_coefficient(angle_outskull, random_density, random_speed, skull_density, skull_speed);
			RC_water_skull = out_rc;
			TC_water_skull = 1 - out_rc;
		}


		////////////////////////////////////////////
		//second ray::  skull layer1 -> skull layer2
		///////////////////////////////////////////
		PerRayData_beam second_prd;
		second_prd.isHit = true;


		////////////////////////////////////////////
		//R1::  skull layer1 -> skull layer2 refraction vector
		///////////////////////////////////////////
		float3 Refract1 = calVecRefraction(out_skull_entervector, first_prd.ffnormal, skull_speed, random_speed);
		float3 temp_hit_pos1 = first_prd.hit_pos + 0.1 *Refract1;
		float3 temp_normal1;

		if (!((Refract1.x == 0.0) && (Refract1.y == 0.0) && (Refract1.z == 0.0)))
		{

			////////////////////////////////////////////
			//temp_hit_pos:: to ignore artifacts between skull layers
			///////////////////////////////////////////

			while (second_prd.isHit) {
				Ray refraction1(temp_hit_pos1, Refract1, BEAM_RAY_TYPE, scene_epsilon);
				rtTrace(top_object, refraction1, second_prd);

				if (second_prd.isHit)
				{

					temp_hit_pos1 = second_prd.hit_pos;
					temp_normal1 = second_prd.ffnormal;
				}
			}
		}
		second_prd.isHit = !second_prd.isHit;

		////////////////////////////////////////////
		//in_rc:: skull layer1 -> skull layer2
		///////////////////////////////////////////
		if (second_prd.isHit)
		{
			angle_inskull = getIncidenceAngle(Refract1, temp_normal1);
			in_rc = reflection_coefficient(angle_inskull, skull_density, skull_speed, water_density, water_speed);
			RC_skull_brain = in_rc;
			TC_skull_brain = 1 - in_rc;
		}

		in_out_dist = first_prd.hit_pos - temp_hit_pos1;
		travel_length = magnitude_cu(in_out_dist);
		phase_refraction = _exp(make_hipDoubleComplex(0, k*travel_length));

		PerRayData_beam reflect_prd;
		reflect_prd.isHit = false;
		float3 R2 = calVecRefraction(Refract1, temp_normal1, water_speed, skull_speed);

		float3 temp_hit_pos2;
		float3 temp_normal2;
		if (!((R2.x == 0) && (R2.y == 0) && (R2.z == 0)))
		{

			////////////////////////////////////////////
			//reflect_prd:: skull layer2 -> skull layer1 reflect ray
			////////////////////////////////////////////
			float3 reflection_start = temp_hit_pos1 - 0.1 * Refract1;
			float3 in_reflection = calVecReflection(Refract1, temp_normal1);


			////////////////////////////////////////////
			//temp_prd:: keep rt until layer 1 (ignore artifacts between skull layers)
			///////////////////////////////////////////
			PerRayData_beam reflection_prd;
			reflection_prd.isHit = true;
			temp_hit_pos2 = reflection_start;

			while (reflection_prd.isHit) {
				Ray reflection(temp_hit_pos2, in_reflection, BEAM_RAY_TYPE, scene_epsilon);
				rtTrace(top_object, reflection, reflection_prd);
				if (reflection_prd.isHit)
				{
					temp_hit_pos2 = reflection_prd.hit_pos;
					temp_normal2 = reflection_prd.ffnormal;
				}
			}

			reflection_prd.isHit = !reflection_prd.isHit;
			if (reflection_prd.isHit) {
				float3 reflection_vector = (temp_hit_pos1 + 35 * in_reflection) - reflection_start;
				float3 reflection_vector_from_innerskull = calVecRefraction(reflection_vector, temp_normal2, random_speed, skull_speed);

				angle_reflection = getIncidenceAngle(reflection_vector, temp_normal2);
				reflect_rc = reflection_coefficient(angle_reflection, skull_density, skull_speed, random_density, random_speed);
				RC_skull_water = reflect_rc;
				TC_skull_water = 1 - reflect_rc;
			}
			in_reflection_dist = temp_hit_pos1 - temp_hit_pos2;
			travel_length_reflection = magnitude_cu(in_reflection_dist);
			phase_reflection = _exp(make_hipDoubleComplex(0, k*travel_length_reflection));

		}

		if (out_rc == 1) {
			test_rc = 1;
		}
		else {

			hipDoubleComplex result = hipCmul(phase_refraction, make_hipDoubleComplex(TC_water_skull, 0));
			result = hipCmul(result, make_hipDoubleComplex(RC_skull_brain, 0));
			result = hipCmul(result, phase_reflection);
			result = hipCmul(result, make_hipDoubleComplex(TC_skull_water, 0));
			test_rc = hipCabs(hipCadd(result, make_hipDoubleComplex(RC_water_skull, 0)));
			
		}

		//rtPrintf("%d ray_origin == : [%f %f %f]\n", idx, ray_origin.x, ray_origin.y, ray_origin.z);
		//rtPrintf("%d == first hit: [%f, %f, %f]\n", idx, first_prd.hit_pos.x, first_prd.hit_pos.y, first_prd.hit_pos.z);
		//rtPrintf("%d == norma1: [%f, %f, %f]\n", idx, first_prd.ffnormal.x, first_prd.ffnormal.y, first_prd.ffnormal.z);
		//rtPrintf("%d == second hit: [%f, %f, %f]\n", idx, temp_hit_pos1.x, temp_hit_pos1.y, temp_hit_pos1.z);
		//rtPrintf("%d == norma2: [%f, %f, %f]\n", idx, temp_normal1.x, temp_normal1.y, temp_normal1.z);
		//rtPrintf("%d == refr2: [%f, %f, %f]\n", idx, R2.x, R2.y, R2.z);
		//rtPrintf("%d == reflect hit: [%f, %f, %f]\n", idx, temp_hit_pos2.x, temp_hit_pos2.y, temp_hit_pos2.z);
		//rtPrintf("%d == normarefle: [%f, %f, %f]\n", idx, temp_normal2.x, temp_normal2.y, temp_normal2.z);
		//rtPrintf("%d ==  out_rc %f\n", idx, out_rc);
		//rtPrintf("%d == test_rc %f\n", idx, test_rc);


		out_rc_buffer[launch_index] = out_rc;
		test_rc_buffer[launch_index] = test_rc;
		out_intersection_point[launch_index] = first_prd.hit_pos;
		in_intersection_point[launch_index] = second_prd.hit_pos;
		reflec_in_skull2out[launch_index] = reflect_prd.hit_pos;
	
}


RT_PROGRAM void closest_hit_skull()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;

	prd_beam.ray_direction = ray.direction;
	prd_beam.ffnormal = ffnormal;
	prd_beam.hit_pos = hit_point;
	prd_beam.isHit = true;

}


RT_PROGRAM void any_hit()
{ 

	rtTerminateRay();
	
}

RT_PROGRAM void miss()
{
	
	prd_beam.isHit = false;
	
}
 
RT_PROGRAM void exception()
{
	rtPrintExceptionDetails();
}
